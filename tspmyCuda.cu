
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <climits>
#include <cstdlib>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

#define NMAX 1000

__device__ int calculateTimeToTravel(int* matrix, int from, int to, int N) {
    if (matrix[from * N + to] == 0 || from == to)
        return -1;
    return matrix[from * N + to];
}

__global__ void tspKernel(int *matrix, int N, int *results, int *paths, int ITERATIONS) {  
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= ITERATIONS) return;

    bool visited[NMAX];
    int path[NMAX + 1];
    int iterators[NMAX];
    int total_time = 0;
    int curr = 0;

    for (int i = 0; i < N; i++) {
        visited[i] = false;
        iterators[i] = i;
    }

    visited[0] = true;
    path[0] = 0;

    hiprandState state;
    hiprand_init(clock64(), tid, 0, &state);

    for (int step = 1; step < N; step++) {
        for (int i = N - 1; i > 0; i--) {
            int j = hiprand(&state) % (i + 1);
            int tmp = iterators[i];
            iterators[i] = iterators[j];
            iterators[j] = tmp;
        }

        int min_time = INT_MAX;
        int next_vertex = -1;

        for (int i = 0; i < N; ++i) {
            int neighbour = iterators[i];
            int time = calculateTimeToTravel(matrix, curr, neighbour, N);
            if (time != -1 && !visited[neighbour] && time < min_time) {
                min_time = time;
                next_vertex = neighbour;
            }
        }

        if (next_vertex == -1) {
            results[tid] = INT_MAX;
            return;
        }

        total_time += min_time;
        curr = next_vertex;
        visited[curr] = true;
        path[step] = curr;
    }

    int return_time = calculateTimeToTravel(matrix, curr, 0, N);
    if (return_time == -1) {
        results[tid] = INT_MAX;
        return;
    }

    total_time += return_time;
    path[N] = 0;
    results[tid] = total_time;

    for (int i = 0; i <= N; i++)
        paths[tid * (N + 1) + i] = path[i];
}

void readMatrix(const std::string &filename, std::vector<int> &matrix, int &N) {
    std::ifstream file(filename);
    file >> N;
    matrix.resize(N * N);
    for (int i = 0; i < N * N; ++i)
        file >> matrix[i];
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file>" << std::endl;
        return 1;
    }
    int ITERATIONS = atoi(argv[1]); // Liczba iteracji do wykonania
    int N;
    std::vector<int> matrix;
    readMatrix(argv[2], matrix, N);

    int *d_matrix, *d_results, *d_paths;
    hipMalloc(&d_matrix, N * N * sizeof(int));
    hipMalloc(&d_results, ITERATIONS * sizeof(int));
    hipMalloc(&d_paths, ITERATIONS * (N + 1) * sizeof(int));
    hipMemcpy(d_matrix, matrix.data(), N * N * sizeof(int), hipMemcpyHostToDevice);

    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    std::cout << "Maksymalna liczba wątków w bloku (dla tego GPU): " << maxThreadsPerBlock << std::endl;

    int blockSize = maxThreadsPerBlock;
    int gridSize = (ITERATIONS + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Uruchomienie kernela z maksymalnym blockSize
    tspKernel<<<gridSize, blockSize>>>(d_matrix, N, d_results, d_paths, ITERATIONS);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::vector<int> results(ITERATIONS), paths(ITERATIONS * (N + 1));
    hipMemcpy(results.data(), d_results, ITERATIONS * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(paths.data(), d_paths, ITERATIONS * (N + 1) * sizeof(int), hipMemcpyDeviceToHost);

    int min_time = INT_MAX, best_idx = -1;
    for (int i = 0; i < ITERATIONS; ++i) {
        if (results[i] < min_time) {
            min_time = results[i];
            best_idx = i;
        }
    }

    std::cout << "Najlepsza ścieżka: ";
    for (int i = 0; i <= N; ++i)
        std::cout << paths[best_idx * (N + 1) + i] << " ";
    std::cout << "\nCzas ścieżki: " << min_time << std::endl;

    std::cout << "Czas działania algorytmu (GPU): " << milliseconds / 1000 << " s" << std::endl;

    hipFree(d_matrix);
    hipFree(d_results);
    hipFree(d_paths);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
